#include "hip/hip_runtime.h"
#include "convnet_fft.cuh"

__global__ void kernel_sample()
{
	printf("Block = %d, Thread %d\n", blockIdx.x, threadIdx.x);
}

int main()
{
	kernel_sample<<<2, 3>>>();
	hipDeviceSynchronize();
	return 0;
}
